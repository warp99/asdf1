#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#define GLEW_STATIC
#include "glew.h"
#include "GL/glut.h"
#include "cuda_gl_interop.h"


int window_width = 512;
int window_height = 512;
int mesh_width = 256;
int mesh_height = 256;
float anim = 0.0f;

//mouse
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;


void display();
void mouse (int button, int state, int x, int y);
void motion(int x, int y);
void deleteVBO();
__global__ void createVertices(float4* positions, float time,
								unsigned int mesh_width, unsigned int mesh_height);

GLuint positionsVBO;
struct hipGraphicsResource *positionsVBO_CUDA;

int main(int argc, char **argv)
{
	puts("krok1");
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowPosition(100,100);
	glutInitWindowSize(window_width,window_height);
	glutCreateWindow("Janix is the Mastah!");

	puts ("krok2");

	cudaGLSetGLDevice(0);
/*
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	glTranslatef(0.0, 0.0, translate_z);
	glRotatef(rotate_x, 1.0, 0.0, 0.0);
	glRotatef(rotate_y, 0.0, 1.0, 0.0);
*/
	glutDisplayFunc(display);
	glutMouseFunc(mouse);
	glutMotionFunc(motion);

	glClearColor(0.0, 0.0, 0.0, 1.0);
	
	//GLEW INIT!!!
	GLenum err = glewInit();
	if (GLEW_OK != err)
	{
		fprintf(stderr, "Error: %s\n",
			glewGetErrorString(err));
	}
	fprintf(stdout, "status: using GLEW %s\n",
		glewGetString(GLEW_VERSION));
	//END GLEW!! :)
	glViewport(0, 0, window_width, window_height);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluPerspective(60.0, (GLfloat)window_width / (GLfloat) window_height, 0.1, 10.0);
	

	puts("krok3");
	glGenBuffers(1, &positionsVBO);
	puts("krok4");
	glBindBuffer(GL_ARRAY_BUFFER, positionsVBO);
	puts("krok5");
	unsigned int size = mesh_width * mesh_height * 4 * sizeof(float);
	glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	hipGraphicsGLRegisterBuffer(&positionsVBO_CUDA, 
									positionsVBO,
									cudaGraphicsMapFlagsWriteDiscard);

	

	glutMainLoop();
	deleteVBO();

}

void display()
{
	float4* positions;
	hipGraphicsMapResources(1, &positionsVBO_CUDA, 0);
	size_t num_bytes;
	hipGraphicsResourceGetMappedPointer((void**)&positions, &num_bytes,
											positionsVBO_CUDA);
	dim3 dimBlock(16, 16, 1);
	dim3 dimGrid(mesh_width / dimBlock.x, mesh_height /dimBlock.y, 1);
	createVertices<<<dimGrid, dimBlock>>>(positions, anim, mesh_width, mesh_height);

	hipGraphicsUnmapResources(1, &positionsVBO_CUDA, 0);
	
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	glTranslatef(0.0, 0.0, translate_z);
	glRotatef(rotate_x, 1.0, 0.0, 0.0);
	glRotatef(rotate_y, 0.0, 1.0, 0.0);

	glBindBuffer(GL_ARRAY_BUFFER, positionsVBO);
	glVertexPointer(4, GL_FLOAT, 0, 0);
	glEnableClientState(GL_VERTEX_ARRAY);
	glDrawArrays(GL_POINTS, 0, mesh_width * mesh_height);
	glDisableClientState(GL_VERTEX_ARRAY);
	anim += 0.01f;

	glutSwapBuffers();
	glutPostRedisplay();
}

void deleteVBO()
{
	hipGraphicsUnregisterResource(positionsVBO_CUDA);
	glDeleteBuffers(1, &positionsVBO);
}

void mouse (int button, int state, int x, int y)
{
	if (state == GLUT_DOWN)
	{
		mouse_buttons |= 1<< button;
	}
	else if (state == GLUT_UP) 
	{
		mouse_buttons = 0;
	}

	mouse_old_x = x;
	mouse_old_y = y;
}
void motion(int x, int y)
{
	float dx, dy;
	dx = (float) (x - mouse_old_x);
	dy = (float) (y - mouse_old_y);
	
	if(mouse_buttons & 1){
		rotate_x += dy * 0.2f;
		rotate_y += dx * 0.2f;
	} else if (mouse_buttons & 4) {
		translate_z += dy * 0.01f;
	}

	mouse_old_x = x;
	mouse_old_y = y;
}


/*
__device__ float3 bodyBodyInteraction(float4 bi, float4 bj, float3 ai)
{
	float3 r;
	r.x = bj.x - bi.x;
	r.y = bj.y - bi.y;
	float odleglosc = r.x * r.x + r.y * r.y;
	ai.x += odleglosc;
	ai.y += odleglosc;
	return ai;

}
*/
/*
__global__ void calculate_forces (void *devX,

positions = bodyBodyInteraction(positions, positions, time)

*/



__global__ void createVertices(float4* positions, float time,
								unsigned int mesh_width, unsigned int mesh_height)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	float u = x / (float) mesh_width;
	float v = y / (float) mesh_height;
	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;
	float d = sqrt(u*u + v*v);
	//float freq = 4.0f;

	float w = sinf(d * time) * cosf(d * time);
	positions[y * mesh_width + x] = make_float4(u, w, v, 1.0f);

}

